#include "cuda_utils.cuh"



// allocate memory on device
hipError_t allocate_memory_on_device_i(void** d_ptr, size_t size)
{
    return hipMalloc((void**) d_ptr, size * sizeof(int));
}

hipError_t allocate_memory_on_device_cv(void** d_ptr, size_t size)
{
    return hipMalloc((void**) d_ptr, size * sizeof(CUSTOMREAL));
}

hipError_t allocate_memory_on_device_bl(void** d_ptr, size_t size)
{
    return hipMalloc((void**) d_ptr, size * sizeof(bool));
}


// device-host shared memory (pinned memory) (maybe unnecessary for CUDA-aware MPI)
hipError_t allocate_memory_on_device_cv_pinned(void** d_ptr, size_t size)
{
    return hipHostMalloc((void**) d_ptr, size * sizeof(CUSTOMREAL));
}


// deallocate memory on device
hipError_t deallocate_memory_on_device_i(int*& d_ptr)
{
    return hipFree(d_ptr);
}

hipError_t deallocate_memory_on_device_cv(CUSTOMREAL*& d_ptr)
{
    return hipFree(d_ptr);
}

hipError_t deallocate_memory_on_device_bl(bool*& d_ptr)
{
    return hipFree(d_ptr);
}


// copy memory from host to device
hipError_t copy_host_to_device_i(int* d_ptr, int* h_ptr, const size_t size)
{
    return hipMemcpy(d_ptr, h_ptr, size * sizeof(int), hipMemcpyHostToDevice);
}

hipError_t copy_host_to_device_cv(CUSTOMREAL* d_ptr, CUSTOMREAL* h_ptr, const size_t size)
{
    return hipMemcpy(d_ptr, h_ptr, size * sizeof(CUSTOMREAL), hipMemcpyHostToDevice);
}

hipError_t copy_host_to_device_bl(bool* d_ptr, bool* h_ptr, const size_t size)
{
    return hipMemcpy(d_ptr, h_ptr, size * sizeof(bool), hipMemcpyHostToDevice);
}

// copy memory from device to host
hipError_t copy_device_to_host_i(int* h_ptr, int* d_ptr, size_t size)
{
    return hipMemcpy(h_ptr, d_ptr, size * sizeof(int), hipMemcpyDeviceToHost);
}
hipError_t copy_device_to_host_cv(CUSTOMREAL* h_ptr, CUSTOMREAL* d_ptr, size_t size)
{
    return hipMemcpy(h_ptr, d_ptr, size * sizeof(CUSTOMREAL), hipMemcpyDeviceToHost);
}


// allocate and copy to device
hipError_t allocate_and_copy_host_to_device_i(int* d_ptr, int* h_ptr, size_t size)
{
    hipError_t err0 = allocate_memory_on_device_i((void**)&d_ptr, size);
    hipError_t err1 = copy_host_to_device_i(d_ptr, h_ptr, size);

    return err1;
}

hipError_t allocate_and_copy_host_to_device_cv(CUSTOMREAL* d_ptr, CUSTOMREAL* h_ptr, size_t size)
{
    hipError_t err0 = allocate_memory_on_device_cv((void**) &d_ptr, size);
    hipError_t err1 = copy_host_to_device_cv(d_ptr, h_ptr, size);

    return err1;
}

// allocate, flatten and copy from host to device
void flatten_arr_i(int* h_ptr_flattened, std::vector<int*>&h_v, int size_total, int* size_each)
{
    // flatten
    int counter = 0;
    int n_v = h_v.size();

    for (int i = 0; i < n_v; i++) { // levels
        for (int j = 0; j < size_each[i]; j++) {
            h_ptr_flattened[counter] = h_v.at(i)[j];
            counter++;
        }
    }
}

void flatten_arr_cv(CUSTOMREAL* h_ptr_flattened, std::vector<CUSTOMREAL*> &h_v, int size_total, int* size_each)
{
    // flatten
    int counter = 0;
    int n_v = h_v.size();

    for (int i = 0; i < n_v; i++) { // levels
        for (int j = 0; j < size_each[i]; j++) {
            h_ptr_flattened[counter] = h_v.at(i)[j];
            counter++;
        }
    }
}

hipError_t allocate_and_copy_host_to_device_flattened_i(int* d_ptr, std::vector<int*>& vh, int size_total, int* size_each){
    // flatten
    int* h_ptr_flattened = new int[size_total];
    flatten_arr_i(h_ptr_flattened, vh, size_total, size_each);

    // allocate and copy
    hipError_t err = allocate_and_copy_host_to_device_i(d_ptr, h_ptr_flattened, size_total);

    // free
    delete[] h_ptr_flattened;

    return err;
}

hipError_t allocate_and_copy_host_to_device_flattened_cv(CUSTOMREAL* d_ptr, std::vector<CUSTOMREAL*>& vh, int size_total, int* size_each){
    // flatten
    CUSTOMREAL* h_ptr_flattened = new CUSTOMREAL[size_total];
    flatten_arr_cv(h_ptr_flattened, vh, size_total, size_each);

    // allocate and copy
    hipError_t err = allocate_and_copy_host_to_device_cv(d_ptr, h_ptr_flattened, size_total);

    // free
    delete[] h_ptr_flattened;

    return err;
}

