#include "hip/hip_runtime.h"
#include "iterator_wrapper.cuh"

__device__ const CUSTOMREAL PLUS = 1.0f;
__device__ const CUSTOMREAL MINUS = -1.0f;
__device__ const CUSTOMREAL v_eps = 1e-12;

__device__ const CUSTOMREAL _0_5_CR   = 0.5f;
__device__ const CUSTOMREAL _1_CR     = 1.0f;
__device__ const CUSTOMREAL _2_CR     = 2.0f;
__device__ const CUSTOMREAL _3_CR     = 3.0f;
__device__ const CUSTOMREAL _4_CR     = 4.0f;

__device__ CUSTOMREAL my_square_cu(CUSTOMREAL const& x) {
    return x*x;
}

__device__ CUSTOMREAL calc_stencil_1st(CUSTOMREAL const& a, CUSTOMREAL const& b, CUSTOMREAL const& Dinv){
    return Dinv*(a-b);
}

__device__ CUSTOMREAL calc_stencil_3rd(CUSTOMREAL const& a, CUSTOMREAL const& b, CUSTOMREAL const& c, CUSTOMREAL const& d, CUSTOMREAL const& Dinv_half, CUSTOMREAL const& sign){
    CUSTOMREAL tmp1 = v_eps + my_square_cu(a-_2_CR*b+c);
    CUSTOMREAL tmp2 = v_eps + my_square_cu(d-_2_CR*a+b);
    CUSTOMREAL ww   = _1_CR/(_1_CR+_2_CR*my_square_cu(tmp1/tmp2));
    return sign*((_1_CR-ww)* (b-d)*Dinv_half + ww*(-_3_CR*a+_4_CR*b-c)*Dinv_half);
}

__device__ CUSTOMREAL cuda_calc_LF_Hamiltonian( \
                                            CUSTOMREAL const& fac_a_, \
                                            CUSTOMREAL const& fac_b_, \
                                            CUSTOMREAL const& fac_c_, \
                                            CUSTOMREAL const& fac_f_, \
                                            CUSTOMREAL const& T0r_, \
                                            CUSTOMREAL const& T0t_, \
                                            CUSTOMREAL const& T0p_, \
                                            CUSTOMREAL const& T0v_, \
                                            CUSTOMREAL& tau_, \
                                            CUSTOMREAL const& pp1, CUSTOMREAL& pp2, \
                                            CUSTOMREAL const& pt1, CUSTOMREAL& pt2, \
                                            CUSTOMREAL const& pr1, CUSTOMREAL& pr2 \
                                            ) {
    // LF Hamiltonian for T = T0 * tau
    return sqrt(
              fac_a_ * my_square_cu(T0r_ * tau_ + T0v_ * (pr1+pr2)/_2_CR) \
    +         fac_b_ * my_square_cu(T0t_ * tau_ + T0v_ * (pt1+pt2)/_2_CR) \
    +         fac_c_ * my_square_cu(T0p_ * tau_ + T0v_ * (pp1+pp2)/_2_CR) \
    -   _2_CR*fac_f_ * (T0t_ * tau_ + T0v_ * (pt1+pt2)/_2_CR) \
                     * (T0p_ * tau_ + T0v_ * (pp1+pp2)/_2_CR) \
    );
}

__global__ void cuda_do_sweep_level_kernel_1st(\
    const int i__j__k__[],\
    const int ip1j__k__[],\
    const int im1j__k__[],\
    const int i__jp1k__[],\
    const int i__jm1k__[],\
    const int i__j__kp1[],\
    const int i__j__km1[],\
    const CUSTOMREAL fac_a[], \
    const CUSTOMREAL fac_b[], \
    const CUSTOMREAL fac_c[], \
    const CUSTOMREAL fac_f[], \
    const CUSTOMREAL T0v[], \
    const CUSTOMREAL T0r[], \
    const CUSTOMREAL T0t[], \
    const CUSTOMREAL T0p[], \
    const CUSTOMREAL fun[], \
    const CUSTOMREAL changed[], \
    CUSTOMREAL tau[], \
    const int loc_I, \
    const int loc_J, \
    const int loc_K, \
    const CUSTOMREAL dr, \
    const CUSTOMREAL dt, \
    const CUSTOMREAL dp, \
    const int n_nodes_this_level, \
    const int i_start \
){

    unsigned int i_node = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
    //unsigned int i_node = (blockIdx.x + blockIdx.y*gridDim.x)*blockDim.x + threadIdx.x;

    if (i_node >= n_nodes_this_level) return;

    i_node += i_start;

    //if (i_node >= loc_I*loc_J*loc_K) return;

    if (changed[i_node] != _1_CR) return;

    CUSTOMREAL sigr = _1_CR*sqrt(fac_a[i_node])*T0v[i_node];
    CUSTOMREAL sigt = _1_CR*sqrt(fac_b[i_node])*T0v[i_node];
    CUSTOMREAL sigp = _1_CR*sqrt(fac_c[i_node])*T0v[i_node];
    CUSTOMREAL coe  = _1_CR/((sigr/dr)+(sigt/dt)+(sigp/dp));

    CUSTOMREAL pp1 = calc_stencil_1st(tau[i__j__k__[i_node]],tau[im1j__k__[i_node]], _1_CR/dp);
    CUSTOMREAL pp2 = calc_stencil_1st(tau[ip1j__k__[i_node]],tau[i__j__k__[i_node]], _1_CR/dp);

    CUSTOMREAL pt1 = calc_stencil_1st(tau[i__j__k__[i_node]],tau[i__jm1k__[i_node]], _1_CR/dt);
    CUSTOMREAL pt2 = calc_stencil_1st(tau[i__jp1k__[i_node]],tau[i__j__k__[i_node]], _1_CR/dt);

    CUSTOMREAL pr1 = calc_stencil_1st(tau[i__j__k__[i_node]],tau[i__j__km1[i_node]], _1_CR/dr);
    CUSTOMREAL pr2 = calc_stencil_1st(tau[i__j__kp1[i_node]],tau[i__j__k__[i_node]], _1_CR/dr);

    // LF Hamiltonian
    CUSTOMREAL Htau = cuda_calc_LF_Hamiltonian(\
                                               fac_a[i_node], \
                                               fac_b[i_node], \
                                               fac_c[i_node], \
                                               fac_f[i_node], \
                                               T0r[i_node], \
                                               T0t[i_node], \
                                               T0p[i_node], \
                                               T0v[i_node], \
                                               tau[i__j__k__[i_node]], \
                                               pp1, pp2, pt1, pt2, pr1, pr2);

    tau[i__j__k__[i_node]] += coe*((fun[i_node] - Htau) \
                                  +(sigr*(pr2-pr1) + sigt*(pt2-pt1) + sigp*(pp2-pp1))/_2_CR);

}

__global__ void cuda_do_sweep_level_kernel_3rd(\
    const int i__j__k__[],\
    const int ip1j__k__[],\
    const int im1j__k__[],\
    const int i__jp1k__[],\
    const int i__jm1k__[],\
    const int i__j__kp1[],\
    const int i__j__km1[],\
    const int ip2j__k__[],\
    const int im2j__k__[],\
    const int i__jp2k__[],\
    const int i__jm2k__[],\
    const int i__j__kp2[],\
    const int i__j__km2[],\
    const CUSTOMREAL fac_a[], \
    const CUSTOMREAL fac_b[], \
    const CUSTOMREAL fac_c[], \
    const CUSTOMREAL fac_f[], \
    const CUSTOMREAL T0v[], \
    const CUSTOMREAL T0r[], \
    const CUSTOMREAL T0t[], \
    const CUSTOMREAL T0p[], \
    const CUSTOMREAL fun[], \
    const CUSTOMREAL changed[], \
    CUSTOMREAL tau[], \
    const int loc_I, \
    const int loc_J, \
    const int loc_K, \
    const CUSTOMREAL dr, \
    const CUSTOMREAL dt, \
    const CUSTOMREAL dp,  \
    const int n_nodes_this_level, \
    const int i_start \
){

    CUSTOMREAL pp1, pp2, pt1, pt2, pr1, pr2;

    unsigned int i_node = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
    //unsigned int i_node = (blockIdx.x + blockIdx.y*gridDim.x)*blockDim.x + threadIdx.x;

    if (i_node >= n_nodes_this_level) return;

    i_node += i_start;
    //if (i_node >= loc_I*loc_J*loc_K) return;


    if (changed[i_node] != _1_CR) return;

    int k = i__j__k__[i_node]/(loc_I*loc_J);
    int j = (i__j__k__[i_node] - k*loc_I*loc_J)/loc_I;
    int i = i__j__k__[i_node] - k*loc_I*loc_J - j*loc_I;


    CUSTOMREAL DRinv = _1_CR/dr;
    CUSTOMREAL DTinv = _1_CR/dt;
    CUSTOMREAL DPinv = _1_CR/dp;
    CUSTOMREAL DRinv_half = DRinv*_0_5_CR;
    CUSTOMREAL DTinv_half = DTinv*_0_5_CR;
    CUSTOMREAL DPinv_half = DPinv*_0_5_CR;

    CUSTOMREAL sigr = _1_CR*sqrt(fac_a[i_node])*T0v[i_node];
    CUSTOMREAL sigt = _1_CR*sqrt(fac_b[i_node])*T0v[i_node];
    CUSTOMREAL sigp = _1_CR*sqrt(fac_c[i_node])*T0v[i_node];
    CUSTOMREAL coe  = _1_CR/((sigr/dr)+(sigt/dt)+(sigp/dp));

    // direction p
    if (i == 1) {
        pp1 = calc_stencil_1st(tau[i__j__k__[i_node]],tau[im1j__k__[i_node]],DPinv);
        pp2 = calc_stencil_3rd(tau[i__j__k__[i_node]],tau[ip1j__k__[i_node]],tau[ip2j__k__[i_node]],tau[im1j__k__[i_node]],DPinv_half, PLUS);
    } else if (i == loc_I-2) {
        pp1 = calc_stencil_3rd(tau[i__j__k__[i_node]],tau[im1j__k__[i_node]],tau[im2j__k__[i_node]],tau[ip1j__k__[i_node]],DPinv_half, MINUS);
        pp2 = calc_stencil_1st(tau[ip1j__k__[i_node]],tau[i__j__k__[i_node]],DPinv);
    } else {
        pp1 = calc_stencil_3rd(tau[i__j__k__[i_node]],tau[im1j__k__[i_node]],tau[im2j__k__[i_node]],tau[ip1j__k__[i_node]],DPinv_half, MINUS);
        pp2 = calc_stencil_3rd(tau[i__j__k__[i_node]],tau[ip1j__k__[i_node]],tau[ip2j__k__[i_node]],tau[im1j__k__[i_node]],DPinv_half, PLUS);
    }

    // direction t
    if (j == 1) {
        pt1 = calc_stencil_1st(tau[i__j__k__[i_node]],tau[i__jm1k__[i_node]],DTinv);
        pt2 = calc_stencil_3rd(tau[i__j__k__[i_node]],tau[i__jp1k__[i_node]],tau[i__jp2k__[i_node]],tau[i__jm1k__[i_node]],DTinv_half, PLUS);
    } else if (j == loc_J-2) {
        pt1 = calc_stencil_3rd(tau[i__j__k__[i_node]],tau[i__jm1k__[i_node]],tau[i__jm2k__[i_node]],tau[i__jp1k__[i_node]],DTinv_half, MINUS);
        pt2 = calc_stencil_1st(tau[i__jp1k__[i_node]],tau[i__j__k__[i_node]],DTinv);
    } else {
        pt1 = calc_stencil_3rd(tau[i__j__k__[i_node]],tau[i__jm1k__[i_node]],tau[i__jm2k__[i_node]],tau[i__jp1k__[i_node]],DTinv_half, MINUS);
        pt2 = calc_stencil_3rd(tau[i__j__k__[i_node]],tau[i__jp1k__[i_node]],tau[i__jp2k__[i_node]],tau[i__jm1k__[i_node]],DTinv_half, PLUS);
    }

    // direction r
    if (k == 1) {
        pr1 = calc_stencil_1st(tau[i__j__k__[i_node]],tau[i__j__km1[i_node]],DRinv);
        pr2 = calc_stencil_3rd(tau[i__j__k__[i_node]],tau[i__j__kp1[i_node]],tau[i__j__kp2[i_node]],tau[i__j__km1[i_node]],DRinv_half, PLUS);
    } else if (k == loc_K-2) {
        pr1 = calc_stencil_3rd(tau[i__j__k__[i_node]],tau[i__j__km1[i_node]],tau[i__j__km2[i_node]],tau[i__j__kp1[i_node]],DRinv_half, MINUS);
        pr2 = calc_stencil_1st(tau[i__j__kp1[i_node]],tau[i__j__k__[i_node]],DRinv);
    } else {
        pr1 = calc_stencil_3rd(tau[i__j__k__[i_node]],tau[i__j__km1[i_node]],tau[i__j__km2[i_node]],tau[i__j__kp1[i_node]],DRinv_half, MINUS);
        pr2 = calc_stencil_3rd(tau[i__j__k__[i_node]],tau[i__j__kp1[i_node]],tau[i__j__kp2[i_node]],tau[i__j__km1[i_node]],DRinv_half, PLUS);
    }

    CUSTOMREAL Htau = cuda_calc_LF_Hamiltonian(\
                                               fac_a[i_node], \
                                               fac_b[i_node], \
                                               fac_c[i_node], \
                                               fac_f[i_node], \
                                               T0r[i_node], \
                                               T0t[i_node], \
                                               T0p[i_node], \
                                               T0v[i_node], \
                                               tau[i__j__k__[i_node]], \
                                               pp1, pp2, pt1, pt2, pr1, pr2);

    tau[i__j__k__[i_node]] += coe*((fun[i_node] - Htau) \
                                  +(sigr*(pr2-pr1) + sigt*(pt2-pt1) + sigp*(pp2-pp1))/_2_CR);


}


void initialize_sweep_params(Grid_on_device* grid_dv){

    // check the numBlockPerSm and set the block size accordingly
    int numBlocksPerSm = 0;
    int block_size = CUDA_SWEEPING_BLOCK_SIZE;

    int device;
    hipGetDevice(&device);

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device);
    if(grid_dv->if_3rd_order)
        hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSm, cuda_do_sweep_level_kernel_3rd, CUDA_SWEEPING_BLOCK_SIZE, 0);
    else
        hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSm, cuda_do_sweep_level_kernel_1st, CUDA_SWEEPING_BLOCK_SIZE, 0);

    int max_cooperative_blocks = deviceProp.multiProcessorCount*numBlocksPerSm;

    grid_dv->threads_sweep_host = dim3(block_size, 1, 1);
    grid_dv->grid_sweep_host = dim3(max_cooperative_blocks, 1, 1);

    // spawn streams
    //grid_dv->level_streams = (hipStream_t*)malloc(CUDA_MAX_NUM_STREAMS*sizeof(hipStream_t));
    //for (int i = 0; i < CUDA_MAX_NUM_STREAMS; i++) {
    grid_dv->level_streams = (hipStream_t*)malloc(grid_dv->n_levels_host*sizeof(hipStream_t));
    for (int i = 0; i < grid_dv->n_levels_host; i++) {
        //hipStreamCreate(&(grid_dv->level_streams[i]));
        // add null
        //hipStreamCreateWithFlags(&(grid_dv->level_streams[i]), hipStreamNonBlocking);
        grid_dv->level_streams[i] = nullptr;

    }


}


void finalize_sweep_params(Grid_on_device* grid_on_dv){
    // destroy streams
    //for (int i = 0; i < CUDA_MAX_NUM_STREAMS; i++) {
    //for (int i = 0; i < grid_on_dv->n_levels_host; i++) {
    //    hipStreamDestroy(grid_on_dv->level_streams[i]);
    //}

    free(grid_on_dv->level_streams);
}


void run_kernel(Grid_on_device* grid_dv, int const& iswp, int& i_node_offset, int const& i_level, \
                dim3& grid_each, dim3& threads_each, int& n_nodes_this_level){

        int id_stream = i_level;// % CUDA_MAX_NUM_STREAMS;

        if (grid_dv->if_3rd_order) {
           if (iswp == 0){
                void *kernelArgs[]{\
                    &(grid_dv->vv_i__j__k___0), \
                    &(grid_dv->vv_ip1j__k___0), \
                    &(grid_dv->vv_im1j__k___0), \
                    &(grid_dv->vv_i__jp1k___0), \
                    &(grid_dv->vv_i__jm1k___0), \
                    &(grid_dv->vv_i__j__kp1_0), \
                    &(grid_dv->vv_i__j__km1_0), \
                    &(grid_dv->vv_ip2j__k___0), \
                    &(grid_dv->vv_im2j__k___0), \
                    &(grid_dv->vv_i__jp2k___0), \
                    &(grid_dv->vv_i__jm2k___0), \
                    &(grid_dv->vv_i__j__kp2_0), \
                    &(grid_dv->vv_i__j__km2_0), \
                    &(grid_dv->vv_fac_a_0    ), \
                    &(grid_dv->vv_fac_b_0    ), \
                    &(grid_dv->vv_fac_c_0    ), \
                    &(grid_dv->vv_fac_f_0    ), \
                    &(grid_dv->vv_T0v_0      ), \
                    &(grid_dv->vv_T0r_0      ), \
                    &(grid_dv->vv_T0t_0      ), \
                    &(grid_dv->vv_T0p_0      ), \
                    &(grid_dv->vv_fun_0      ), \
                    &(grid_dv->vv_change_0   ), \
                    &(grid_dv->tau), \
                    &(grid_dv->loc_I_host), \
                    &(grid_dv->loc_J_host), \
                    &(grid_dv->loc_K_host), \
                    &(grid_dv->dr_host), \
                    &(grid_dv->dt_host), \
                    &(grid_dv->dp_host), \
                    &n_nodes_this_level, \
                    &i_node_offset \
                };
                print_CUDA_error_if_any(hipLaunchKernel((void*) cuda_do_sweep_level_kernel_3rd, grid_each, threads_each, kernelArgs, 0, grid_dv->level_streams[id_stream]), 30001);

            } else if (iswp == 1){
                void* kernelArgs[]{\
                    &(grid_dv->vv_i__j__k___1), \
                    &(grid_dv->vv_i__jp1k___1), \
                    &(grid_dv->vv_i__jm1k___1), \
                    &(grid_dv->vv_i__j__kp1_1), \
                    &(grid_dv->vv_i__j__km1_1), \
                    &(grid_dv->vv_ip1j__k___1), \
                    &(grid_dv->vv_im1j__k___1), \
                    &(grid_dv->vv_ip2j__k___1), \
                    &(grid_dv->vv_im2j__k___1), \
                    &(grid_dv->vv_i__jp2k___1), \
                    &(grid_dv->vv_i__jm2k___1), \
                    &(grid_dv->vv_i__j__kp2_1), \
                    &(grid_dv->vv_i__j__km2_1), \
                    &(grid_dv->vv_fac_a_1    ), \
                    &(grid_dv->vv_fac_b_1    ), \
                    &(grid_dv->vv_fac_c_1    ), \
                    &(grid_dv->vv_fac_f_1    ), \
                    &(grid_dv->vv_T0v_1      ), \
                    &(grid_dv->vv_T0r_1      ), \
                    &(grid_dv->vv_T0t_1      ), \
                    &(grid_dv->vv_T0p_1      ), \
                    &(grid_dv->vv_fun_1      ), \
                    &(grid_dv->vv_change_1   ), \
                    &(grid_dv->tau), \
                    &(grid_dv->loc_I_host), \
                    &(grid_dv->loc_J_host), \
                    &(grid_dv->loc_K_host), \
                    &(grid_dv->dr_host), \
                    &(grid_dv->dt_host), \
                    &(grid_dv->dp_host), \
                    &n_nodes_this_level, \
                    &i_node_offset \
                };
                print_CUDA_error_if_any(hipLaunchKernel((void*) cuda_do_sweep_level_kernel_3rd, grid_each, threads_each, kernelArgs, 0, grid_dv->level_streams[id_stream]), 30001);

            } else if (iswp == 2){
                void* kernelArgs[]{\
                    &(grid_dv->vv_i__j__k___2), \
                    &(grid_dv->vv_i__j__kp1_2), \
                    &(grid_dv->vv_i__j__km1_2), \
                    &(grid_dv->vv_ip1j__k___2), \
                    &(grid_dv->vv_im1j__k___2), \
                    &(grid_dv->vv_i__jp1k___2), \
                    &(grid_dv->vv_i__jm1k___2), \
                    &(grid_dv->vv_ip2j__k___2), \
                    &(grid_dv->vv_im2j__k___2), \
                    &(grid_dv->vv_i__jp2k___2), \
                    &(grid_dv->vv_i__jm2k___2), \
                    &(grid_dv->vv_i__j__kp2_2), \
                    &(grid_dv->vv_i__j__km2_2), \
                    &(grid_dv->vv_fac_a_2), \
                    &(grid_dv->vv_fac_b_2), \
                    &(grid_dv->vv_fac_c_2), \
                    &(grid_dv->vv_fac_f_2), \
                    &(grid_dv->vv_T0v_2), \
                    &(grid_dv->vv_T0r_2), \
                    &(grid_dv->vv_T0t_2), \
                    &(grid_dv->vv_T0p_2), \
                    &(grid_dv->vv_fun_2), \
                    &(grid_dv->vv_change_2), \
                    &(grid_dv->tau), \
                    &(grid_dv->loc_I_host), \
                    &(grid_dv->loc_J_host), \
                    &(grid_dv->loc_K_host), \
                    &(grid_dv->dr_host), \
                    &(grid_dv->dt_host), \
                    &(grid_dv->dp_host), \
                    &n_nodes_this_level, \
                    &i_node_offset \
                };
                print_CUDA_error_if_any(hipLaunchKernel((void*) cuda_do_sweep_level_kernel_3rd, grid_each, threads_each, kernelArgs, 0, grid_dv->level_streams[id_stream]), 30001);

            } else if (iswp == 3){
                void* kernelArgs[]{\
                    &(grid_dv->vv_i__j__k___3), \
                    &(grid_dv->vv_ip1j__k___3), \
                    &(grid_dv->vv_im1j__k___3), \
                    &(grid_dv->vv_i__jp1k___3), \
                    &(grid_dv->vv_i__jm1k___3), \
                    &(grid_dv->vv_i__j__kp1_3), \
                    &(grid_dv->vv_i__j__km1_3), \
                    &(grid_dv->vv_ip2j__k___3), \
                    &(grid_dv->vv_im2j__k___3), \
                    &(grid_dv->vv_i__jp2k___3), \
                    &(grid_dv->vv_i__jm2k___3), \
                    &(grid_dv->vv_i__j__kp2_3), \
                    &(grid_dv->vv_i__j__km2_3), \
                    &(grid_dv->vv_fac_a_3), \
                    &(grid_dv->vv_fac_b_3), \
                    &(grid_dv->vv_fac_c_3), \
                    &(grid_dv->vv_fac_f_3), \
                    &(grid_dv->vv_T0v_3), \
                    &(grid_dv->vv_T0r_3), \
                    &(grid_dv->vv_T0t_3), \
                    &(grid_dv->vv_T0p_3), \
                    &(grid_dv->vv_fun_3), \
                    &(grid_dv->vv_change_3), \
                    &(grid_dv->tau), \
                    &(grid_dv->loc_I_host), \
                    &(grid_dv->loc_J_host), \
                    &(grid_dv->loc_K_host), \
                    &(grid_dv->dr_host), \
                    &(grid_dv->dt_host), \
                    &(grid_dv->dp_host), \
                    &n_nodes_this_level, \
                    &i_node_offset \
                };
                print_CUDA_error_if_any(hipLaunchKernel((void*) cuda_do_sweep_level_kernel_3rd, grid_each, threads_each, kernelArgs, 0, grid_dv->level_streams[id_stream]), 30001);

            } else if (iswp == 4){
                void* kernelArgs[]{\
                    &(grid_dv->vv_i__j__k___4), \
                    &(grid_dv->vv_ip1j__k___4), \
                    &(grid_dv->vv_im1j__k___4), \
                    &(grid_dv->vv_i__jp1k___4), \
                    &(grid_dv->vv_i__jm1k___4), \
                    &(grid_dv->vv_i__j__kp1_4), \
                    &(grid_dv->vv_i__j__km1_4), \
                    &(grid_dv->vv_ip2j__k___4), \
                    &(grid_dv->vv_im2j__k___4), \
                    &(grid_dv->vv_i__jp2k___4), \
                    &(grid_dv->vv_i__jm2k___4), \
                    &(grid_dv->vv_i__j__kp2_4), \
                    &(grid_dv->vv_i__j__km2_4), \
                    &(grid_dv->vv_fac_a_4), \
                    &(grid_dv->vv_fac_b_4), \
                    &(grid_dv->vv_fac_c_4), \
                    &(grid_dv->vv_fac_f_4), \
                    &(grid_dv->vv_T0v_4), \
                    &(grid_dv->vv_T0r_4), \
                    &(grid_dv->vv_T0t_4), \
                    &(grid_dv->vv_T0p_4), \
                    &(grid_dv->vv_fun_4), \
                    &(grid_dv->vv_change_4), \
                    &(grid_dv->tau), \
                    &(grid_dv->loc_I_host), \
                    &(grid_dv->loc_J_host), \
                    &(grid_dv->loc_K_host), \
                    &(grid_dv->dr_host), \
                    &(grid_dv->dt_host), \
                    &(grid_dv->dp_host), \
                    &n_nodes_this_level, \
                    &i_node_offset \
                };
                print_CUDA_error_if_any(hipLaunchKernel((void*) cuda_do_sweep_level_kernel_3rd, grid_each, threads_each, kernelArgs, 0, grid_dv->level_streams[id_stream]), 30001);

            } else if (iswp == 5) {
                void* kernelArgs[]{\
                    &(grid_dv->vv_i__j__k___5), \
                    &(grid_dv->vv_ip1j__k___5), \
                    &(grid_dv->vv_im1j__k___5), \
                    &(grid_dv->vv_i__jp1k___5), \
                    &(grid_dv->vv_i__jm1k___5), \
                    &(grid_dv->vv_i__j__kp1_5), \
                    &(grid_dv->vv_i__j__km1_5), \
                    &(grid_dv->vv_ip2j__k___5), \
                    &(grid_dv->vv_im2j__k___5), \
                    &(grid_dv->vv_i__jp2k___5), \
                    &(grid_dv->vv_i__jm2k___5), \
                    &(grid_dv->vv_i__j__kp2_5), \
                    &(grid_dv->vv_i__j__km2_5), \
                    &(grid_dv->vv_fac_a_5), \
                    &(grid_dv->vv_fac_b_5), \
                    &(grid_dv->vv_fac_c_5), \
                    &(grid_dv->vv_fac_f_5), \
                    &(grid_dv->vv_T0v_5), \
                    &(grid_dv->vv_T0r_5), \
                    &(grid_dv->vv_T0t_5), \
                    &(grid_dv->vv_T0p_5), \
                    &(grid_dv->vv_fun_5), \
                    &(grid_dv->vv_change_5), \
                    &(grid_dv->tau), \
                    &(grid_dv->loc_I_host), \
                    &(grid_dv->loc_J_host), \
                    &(grid_dv->loc_K_host), \
                    &(grid_dv->dr_host), \
                    &(grid_dv->dt_host), \
                    &(grid_dv->dp_host), \
                    &n_nodes_this_level, \
                    &i_node_offset \
                };
                print_CUDA_error_if_any(hipLaunchKernel((void*) cuda_do_sweep_level_kernel_3rd, grid_each, threads_each, kernelArgs, 0, grid_dv->level_streams[id_stream]), 30001);

            } else if (iswp == 6) {
                void* kernelArgs[]{\
                    &(grid_dv->vv_i__j__k___6), \
                    &(grid_dv->vv_ip1j__k___6), \
                    &(grid_dv->vv_im1j__k___6), \
                    &(grid_dv->vv_i__jp1k___6), \
                    &(grid_dv->vv_i__jm1k___6), \
                    &(grid_dv->vv_i__j__kp1_6), \
                    &(grid_dv->vv_i__j__km1_6), \
                    &(grid_dv->vv_ip2j__k___6), \
                    &(grid_dv->vv_im2j__k___6), \
                    &(grid_dv->vv_i__jp2k___6), \
                    &(grid_dv->vv_i__jm2k___6), \
                    &(grid_dv->vv_i__j__kp2_6), \
                    &(grid_dv->vv_i__j__km2_6), \
                    &(grid_dv->vv_fac_a_6), \
                    &(grid_dv->vv_fac_b_6), \
                    &(grid_dv->vv_fac_c_6), \
                    &(grid_dv->vv_fac_f_6), \
                    &(grid_dv->vv_T0v_6), \
                    &(grid_dv->vv_T0r_6), \
                    &(grid_dv->vv_T0t_6), \
                    &(grid_dv->vv_T0p_6), \
                    &(grid_dv->vv_fun_6), \
                    &(grid_dv->vv_change_6), \
                    &(grid_dv->tau), \
                    &(grid_dv->loc_I_host), \
                    &(grid_dv->loc_J_host), \
                    &(grid_dv->loc_K_host), \
                    &(grid_dv->dr_host), \
                    &(grid_dv->dt_host), \
                    &(grid_dv->dp_host), \
                    &n_nodes_this_level, \
                    &i_node_offset \
                };
                print_CUDA_error_if_any(hipLaunchKernel((void*) cuda_do_sweep_level_kernel_3rd, grid_each, threads_each, kernelArgs, 0, grid_dv->level_streams[id_stream]), 30001);

            } else {
                void* kernelArgs[]{\
                    &(grid_dv->vv_i__j__k___7), \
                    &(grid_dv->vv_ip1j__k___7), \
                    &(grid_dv->vv_im1j__k___7), \
                    &(grid_dv->vv_i__jp1k___7), \
                    &(grid_dv->vv_i__jm1k___7), \
                    &(grid_dv->vv_i__j__kp1_7), \
                    &(grid_dv->vv_i__j__km1_7), \
                    &(grid_dv->vv_ip2j__k___7), \
                    &(grid_dv->vv_im2j__k___7), \
                    &(grid_dv->vv_i__jp2k___7), \
                    &(grid_dv->vv_i__jm2k___7), \
                    &(grid_dv->vv_i__j__kp2_7), \
                    &(grid_dv->vv_i__j__km2_7), \
                    &(grid_dv->vv_fac_a_7), \
                    &(grid_dv->vv_fac_b_7), \
                    &(grid_dv->vv_fac_c_7), \
                    &(grid_dv->vv_fac_f_7), \
                    &(grid_dv->vv_T0v_7), \
                    &(grid_dv->vv_T0r_7), \
                    &(grid_dv->vv_T0t_7), \
                    &(grid_dv->vv_T0p_7), \
                    &(grid_dv->vv_fun_7), \
                    &(grid_dv->vv_change_7), \
                    &(grid_dv->tau), \
                    &(grid_dv->loc_I_host), \
                    &(grid_dv->loc_J_host), \
                    &(grid_dv->loc_K_host), \
                    &(grid_dv->dr_host), \
                    &(grid_dv->dt_host), \
                    &(grid_dv->dp_host), \
                    &n_nodes_this_level, \
                    &i_node_offset \
                };
                print_CUDA_error_if_any(hipLaunchKernel((void*) cuda_do_sweep_level_kernel_3rd, grid_each, threads_each, kernelArgs, 0, grid_dv->level_streams[id_stream]), 30001);

            }
        } else { // 1st order
            if (iswp == 0){
                void* kernelArgs[]{\
                    &(grid_dv->vv_i__j__k___0), \
                    &(grid_dv->vv_ip1j__k___0), \
                    &(grid_dv->vv_im1j__k___0), \
                    &(grid_dv->vv_i__jp1k___0), \
                    &(grid_dv->vv_i__jm1k___0), \
                    &(grid_dv->vv_i__j__kp1_0), \
                    &(grid_dv->vv_i__j__km1_0), \
                    &(grid_dv->vv_fac_a_0), \
                    &(grid_dv->vv_fac_b_0), \
                    &(grid_dv->vv_fac_c_0), \
                    &(grid_dv->vv_fac_f_0), \
                    &(grid_dv->vv_T0v_0), \
                    &(grid_dv->vv_T0r_0), \
                    &(grid_dv->vv_T0t_0), \
                    &(grid_dv->vv_T0p_0), \
                    &(grid_dv->vv_fun_0), \
                    &(grid_dv->vv_change_0), \
                    &(grid_dv->tau), \
                    &(grid_dv->loc_I_host), \
                    &(grid_dv->loc_J_host), \
                    &(grid_dv->loc_K_host), \
                    &(grid_dv->dr_host), \
                    &(grid_dv->dt_host), \
                    &(grid_dv->dp_host), \
                    &n_nodes_this_level, \
                    &i_node_offset \
                };
                print_CUDA_error_if_any(hipLaunchKernel((void*) cuda_do_sweep_level_kernel_1st, grid_each, threads_each, kernelArgs, 0, grid_dv->level_streams[id_stream]), 30000);

            } else if (iswp == 1){
                void* kernelArgs[]{\
                    &(grid_dv->vv_i__j__k___1), \
                    &(grid_dv->vv_i__jp1k___1), \
                    &(grid_dv->vv_i__jm1k___1), \
                    &(grid_dv->vv_i__j__kp1_1), \
                    &(grid_dv->vv_i__j__km1_1), \
                    &(grid_dv->vv_ip1j__k___1), \
                    &(grid_dv->vv_im1j__k___1), \
                    &(grid_dv->vv_fac_a_1), \
                    &(grid_dv->vv_fac_b_1), \
                    &(grid_dv->vv_fac_c_1), \
                    &(grid_dv->vv_fac_f_1), \
                    &(grid_dv->vv_T0v_1), \
                    &(grid_dv->vv_T0r_1), \
                    &(grid_dv->vv_T0t_1), \
                    &(grid_dv->vv_T0p_1), \
                    &(grid_dv->vv_fun_1), \
                    &(grid_dv->vv_change_1), \
                    &(grid_dv->tau), \
                    &(grid_dv->loc_I_host), \
                    &(grid_dv->loc_J_host), \
                    &(grid_dv->loc_K_host), \
                    &(grid_dv->dr_host), \
                    &(grid_dv->dt_host), \
                    &(grid_dv->dp_host), \
                    &n_nodes_this_level, \
                    &i_node_offset \
                };
                print_CUDA_error_if_any(hipLaunchKernel((void*) cuda_do_sweep_level_kernel_1st, grid_each, threads_each, kernelArgs, 0, grid_dv->level_streams[id_stream]), 30001);

            } else if (iswp == 2){
                void* kernelArgs[]{\
                    &(grid_dv->vv_i__j__k___2), \
                    &(grid_dv->vv_i__j__kp1_2), \
                    &(grid_dv->vv_i__j__km1_2), \
                    &(grid_dv->vv_ip1j__k___2), \
                    &(grid_dv->vv_im1j__k___2), \
                    &(grid_dv->vv_i__jp1k___2), \
                    &(grid_dv->vv_i__jm1k___2), \
                    &(grid_dv->vv_fac_a_2), \
                    &(grid_dv->vv_fac_b_2), \
                    &(grid_dv->vv_fac_c_2), \
                    &(grid_dv->vv_fac_f_2), \
                    &(grid_dv->vv_T0v_2), \
                    &(grid_dv->vv_T0r_2), \
                    &(grid_dv->vv_T0t_2), \
                    &(grid_dv->vv_T0p_2), \
                    &(grid_dv->vv_fun_2), \
                    &(grid_dv->vv_change_2), \
                    &(grid_dv->tau), \
                    &(grid_dv->loc_I_host), \
                    &(grid_dv->loc_J_host), \
                    &(grid_dv->loc_K_host), \
                    &(grid_dv->dr_host), \
                    &(grid_dv->dt_host), \
                    &(grid_dv->dp_host), \
                    &n_nodes_this_level, \
                    &i_node_offset \
                };
                print_CUDA_error_if_any(hipLaunchKernel((void*) cuda_do_sweep_level_kernel_1st, grid_each, threads_each, kernelArgs, 0, grid_dv->level_streams[id_stream]), 30002);

            } else if (iswp == 3){
                void* kernelArgs[]{\
                    &(grid_dv->vv_i__j__k___3), \
                    &(grid_dv->vv_ip1j__k___3), \
                    &(grid_dv->vv_im1j__k___3), \
                    &(grid_dv->vv_i__jp1k___3), \
                    &(grid_dv->vv_i__jm1k___3), \
                    &(grid_dv->vv_i__j__kp1_3), \
                    &(grid_dv->vv_i__j__km1_3), \
                    &(grid_dv->vv_fac_a_3), \
                    &(grid_dv->vv_fac_b_3), \
                    &(grid_dv->vv_fac_c_3), \
                    &(grid_dv->vv_fac_f_3), \
                    &(grid_dv->vv_T0v_3), \
                    &(grid_dv->vv_T0r_3), \
                    &(grid_dv->vv_T0t_3), \
                    &(grid_dv->vv_T0p_3), \
                    &(grid_dv->vv_fun_3), \
                    &(grid_dv->vv_change_3), \
                    &(grid_dv->tau), \
                    &(grid_dv->loc_I_host), \
                    &(grid_dv->loc_J_host), \
                    &(grid_dv->loc_K_host), \
                    &(grid_dv->dr_host), \
                    &(grid_dv->dt_host), \
                    &(grid_dv->dp_host), \
                    &n_nodes_this_level, \
                    &i_node_offset \
                };
                print_CUDA_error_if_any(hipLaunchKernel((void*) cuda_do_sweep_level_kernel_1st, grid_each, threads_each, kernelArgs, 0, grid_dv->level_streams[id_stream]), 30003);

            } else if (iswp == 4){
                void* kernelArgs[]{\
                    &(grid_dv->vv_i__j__k___4), \
                    &(grid_dv->vv_ip1j__k___4), \
                    &(grid_dv->vv_im1j__k___4), \
                    &(grid_dv->vv_i__jp1k___4), \
                    &(grid_dv->vv_i__jm1k___4), \
                    &(grid_dv->vv_i__j__kp1_4), \
                    &(grid_dv->vv_i__j__km1_4), \
                    &(grid_dv->vv_fac_a_4), \
                    &(grid_dv->vv_fac_b_4), \
                    &(grid_dv->vv_fac_c_4), \
                    &(grid_dv->vv_fac_f_4), \
                    &(grid_dv->vv_T0v_4), \
                    &(grid_dv->vv_T0r_4), \
                    &(grid_dv->vv_T0t_4), \
                    &(grid_dv->vv_T0p_4), \
                    &(grid_dv->vv_fun_4), \
                    &(grid_dv->vv_change_4), \
                    &(grid_dv->tau), \
                    &(grid_dv->loc_I_host), \
                    &(grid_dv->loc_J_host), \
                    &(grid_dv->loc_K_host), \
                    &(grid_dv->dr_host), \
                    &(grid_dv->dt_host), \
                    &(grid_dv->dp_host), \
                    &n_nodes_this_level, \
                    &i_node_offset \
                };
                print_CUDA_error_if_any(hipLaunchKernel((void*) cuda_do_sweep_level_kernel_1st, grid_each, threads_each, kernelArgs, 0, grid_dv->level_streams[id_stream]), 30004);

            } else if (iswp == 5) {
                void* kernelArgs[]{\
                    &(grid_dv->vv_i__j__k___5), \
                    &(grid_dv->vv_ip1j__k___5), \
                    &(grid_dv->vv_im1j__k___5), \
                    &(grid_dv->vv_i__jp1k___5), \
                    &(grid_dv->vv_i__jm1k___5), \
                    &(grid_dv->vv_i__j__kp1_5), \
                    &(grid_dv->vv_i__j__km1_5), \
                    &(grid_dv->vv_fac_a_5), \
                    &(grid_dv->vv_fac_b_5), \
                    &(grid_dv->vv_fac_c_5), \
                    &(grid_dv->vv_fac_f_5), \
                    &(grid_dv->vv_T0v_5), \
                    &(grid_dv->vv_T0r_5), \
                    &(grid_dv->vv_T0t_5), \
                    &(grid_dv->vv_T0p_5), \
                    &(grid_dv->vv_fun_5), \
                    &(grid_dv->vv_change_5), \
                    &(grid_dv->tau), \
                    &(grid_dv->loc_I_host), \
                    &(grid_dv->loc_J_host), \
                    &(grid_dv->loc_K_host), \
                    &(grid_dv->dr_host), \
                    &(grid_dv->dt_host), \
                    &(grid_dv->dp_host), \
                    &n_nodes_this_level, \
                    &i_node_offset \
                };
                print_CUDA_error_if_any(hipLaunchKernel((void*) cuda_do_sweep_level_kernel_1st, grid_each, threads_each, kernelArgs, 0, grid_dv->level_streams[id_stream]), 30005);

            } else if (iswp == 6) {
                void* kernelArgs[]{\
                    &(grid_dv->vv_i__j__k___6), \
                    &(grid_dv->vv_ip1j__k___6), \
                    &(grid_dv->vv_im1j__k___6), \
                    &(grid_dv->vv_i__jp1k___6), \
                    &(grid_dv->vv_i__jm1k___6), \
                    &(grid_dv->vv_i__j__kp1_6), \
                    &(grid_dv->vv_i__j__km1_6), \
                    &(grid_dv->vv_fac_a_6), \
                    &(grid_dv->vv_fac_b_6), \
                    &(grid_dv->vv_fac_c_6), \
                    &(grid_dv->vv_fac_f_6), \
                    &(grid_dv->vv_T0v_6), \
                    &(grid_dv->vv_T0r_6), \
                    &(grid_dv->vv_T0t_6), \
                    &(grid_dv->vv_T0p_6), \
                    &(grid_dv->vv_fun_6), \
                    &(grid_dv->vv_change_6), \
                    &(grid_dv->tau), \
                    &(grid_dv->loc_I_host), \
                    &(grid_dv->loc_J_host), \
                    &(grid_dv->loc_K_host), \
                    &(grid_dv->dr_host), \
                    &(grid_dv->dt_host), \
                    &(grid_dv->dp_host), \
                    &n_nodes_this_level, \
                    &i_node_offset \
                };
                print_CUDA_error_if_any(hipLaunchKernel((void*) cuda_do_sweep_level_kernel_1st, grid_each, threads_each, kernelArgs, 0, grid_dv->level_streams[id_stream]), 30006);


            } else {
                void* kernelArgs[]{\
                    &(grid_dv->vv_i__j__k___7), \
                    &(grid_dv->vv_ip1j__k___7), \
                    &(grid_dv->vv_im1j__k___7), \
                    &(grid_dv->vv_i__jp1k___7), \
                    &(grid_dv->vv_i__jm1k___7), \
                    &(grid_dv->vv_i__j__kp1_7), \
                    &(grid_dv->vv_i__j__km1_7), \
                    &(grid_dv->vv_fac_a_7    ), \
                    &(grid_dv->vv_fac_b_7    ), \
                    &(grid_dv->vv_fac_c_7    ), \
                    &(grid_dv->vv_fac_f_7    ), \
                    &(grid_dv->vv_T0v_7      ), \
                    &(grid_dv->vv_T0r_7      ), \
                    &(grid_dv->vv_T0t_7      ), \
                    &(grid_dv->vv_T0p_7      ), \
                    &(grid_dv->vv_fun_7      ), \
                    &(grid_dv->vv_change_7   ), \
                    &(grid_dv->tau), \
                    &(grid_dv->loc_I_host), \
                    &(grid_dv->loc_J_host), \
                    &(grid_dv->loc_K_host), \
                    &(grid_dv->dr_host), \
                    &(grid_dv->dt_host), \
                    &(grid_dv->dp_host), \
                    &n_nodes_this_level, \
                    &i_node_offset \
                };

                print_CUDA_error_if_any(hipLaunchKernel((void*) cuda_do_sweep_level_kernel_1st, grid_each, threads_each, kernelArgs, 0, grid_dv->level_streams[id_stream]), 30007);

            }
        }

        // synchronize all streams
        //print_CUDA_error_if_any(hipStreamSynchronize(grid_dv->level_streams[id_stream]), 30008);
}


// this function calculate all levels of one single sweep direction
void cuda_run_iteration_forward(Grid_on_device* grid_dv, int const& iswp){

    initialize_sweep_params(grid_dv);

    int block_size = CUDA_SWEEPING_BLOCK_SIZE;
    int num_blocks_x, num_blocks_y;
    int actual_end_level = grid_dv->n_levels_host;
    int i_node_offset=0;

    for (size_t i_level = 0; i_level < actual_end_level; i_level++){
        get_block_xy(ceil(grid_dv->n_nodes_on_levels_host[i_level]/block_size+0.5), &num_blocks_x, &num_blocks_y);
        dim3 grid_each(num_blocks_x, num_blocks_y);
        dim3 threads_each(block_size, 1, 1);

        run_kernel(grid_dv, iswp, i_node_offset, i_level, grid_each, threads_each, grid_dv->n_nodes_on_levels_host[i_level]);
        //run_kernel(grid_dv, iswp, i_node_offset, i_level, grid_dv->grid_sweep_host, grid_dv->threads_sweep_host, grid_dv->n_nodes_on_levels_host[i_level]);

        i_node_offset += grid_dv->n_nodes_on_levels_host[i_level];
    }

    finalize_sweep_params(grid_dv);

    // check memory leak
    //print_memory_usage();

}